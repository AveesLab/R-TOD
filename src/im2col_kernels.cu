#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"

extern "C" {
#include "im2col.h"
#include "hip/hip_runtime.h"
}

// src: https://github.com/BVLC/caffe/blob/master/src/caffe/util/im2col.cu
// You may also want to read: https://github.com/BVLC/caffe/blob/master/LICENSE

__global__ void im2col_gpu_kernel(const int n, const float* data_im,
        const int height, const int width, const int ksize,
        const int pad,
        const int stride,
        const int height_col, const int width_col,
        float *data_col) {
    int index = blockIdx.x*blockDim.x+threadIdx.x;
    for(; index < n; index += blockDim.x*gridDim.x){
        int w_out = index % width_col;
        int h_index = index / width_col;
        int h_out = h_index % height_col;
        int channel_in = h_index / height_col;
        int channel_out = channel_in * ksize * ksize;
        int h_in = h_out * stride - pad;
        int w_in = w_out * stride - pad;
        float* data_col_ptr = data_col;
        data_col_ptr += (channel_out * height_col + h_out) * width_col + w_out;
        const float* data_im_ptr = data_im;
        data_im_ptr += (channel_in * height + h_in) * width + w_in;
        for (int i = 0; i < ksize; ++i) {
            for (int j = 0; j < ksize; ++j) {
                int h = h_in + i;
                int w = w_in + j;

                *data_col_ptr = (h >= 0 && w >= 0 && h < height && w < width) ?
                    data_im_ptr[i * width + j] : 0;

                //*data_col_ptr = data_im_ptr[ii * width + jj];

                data_col_ptr += height_col * width_col;
            }
        }
    }
}

void im2col_ongpu(float *im,
         int channels, int height, int width,
         int ksize, int stride, int pad, float *data_col){
    // We are going to launch channels * height_col * width_col kernels, each
    // kernel responsible for copying a single-channel grid.
    int height_col = (height + 2 * pad - ksize) / stride + 1;
    int width_col = (width + 2 * pad - ksize) / stride + 1;
    int num_kernels = channels * height_col * width_col;
    im2col_gpu_kernel<<<(num_kernels+BLOCK-1)/BLOCK,
        BLOCK, 0, get_cuda_stream()>>>(
                num_kernels, im, height, width, ksize, pad,
                stride, height_col,
                width_col, data_col);
}



__global__ void im2col_align_gpu_kernel(const int n, const float* data_im,
    const int height, const int width, const int ksize,
    const int pad,
    const int stride,
    const int height_col, const int width_col,
    float *data_col, const int bit_align)
{
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    for (; index < n; index += blockDim.x*gridDim.x) {
        int w_out = index % width_col;
        int h_index = index / width_col;
        int h_out = h_index % height_col;
        int channel_in = h_index / height_col;
        int channel_out = channel_in * ksize * ksize;
        int h_in = h_out * stride - pad;
        int w_in = w_out * stride - pad;
        float* data_col_ptr = data_col;
        //data_col_ptr += (channel_out * height_col + h_out) * width_col + w_out;
        data_col_ptr += channel_out * bit_align + h_out * width_col + w_out;
        const float* data_im_ptr = data_im;
        data_im_ptr += (channel_in * height + h_in) * width + w_in;
        for (int i = 0; i < ksize; ++i) {
            for (int j = 0; j < ksize; ++j) {
                int h = h_in + i;
                int w = w_in + j;

                *data_col_ptr = (h >= 0 && w >= 0 && h < height && w < width) ?
                    data_im_ptr[i * width + j] : 0;


                //data_col_ptr += height_col * width_col;
                data_col_ptr += bit_align;
            }
        }
    }
}

void im2col_align_ongpu(float *im,
    int channels, int height, int width,
    int ksize, int stride, int pad, float *data_col, int bit_align) {
    // We are going to launch channels * height_col * width_col kernels, each
    // kernel responsible for copying a single-channel grid.
    int height_col = (height + 2 * pad - ksize) / stride + 1;
    int width_col = (width + 2 * pad - ksize) / stride + 1;
    int num_kernels = channels * height_col * width_col;
    im2col_align_gpu_kernel << <(num_kernels + BLOCK - 1) / BLOCK,
        BLOCK, 0, get_cuda_stream() >> >(
            num_kernels, im, height, width, ksize, pad,
            stride, height_col,
            width_col, data_col, bit_align);
}


// --------------------------------

#define WARP_SIZE 32

__global__ void float_to_bit_gpu_kernel(float *src, unsigned char *dst, size_t size)
{
    //size_t dst_size = size / 8 + 1;
    //memset(dst, 0, dst_size);
    //uint32_t bit_mask = __ballot_sync(FULL_MASK, src[i] > 0);
    const int size_aligned = size + (WARP_SIZE - size % WARP_SIZE);

    int index = blockIdx.x*blockDim.x + threadIdx.x;
    float src_val;

    for (; index < size_aligned; index += blockDim.x*gridDim.x)
    {
        if(index < size) src_val = src[index];
        else src_val = 0;
        unsigned int bit_mask = __ballot_sync(0xffffffff, src_val > 0);
        if (threadIdx.x % WARP_SIZE == 0) ((unsigned int*)dst)[index / 32] = bit_mask;
    }
}


void float_to_bit_gpu(float *src, unsigned char *dst, size_t size)
{
    const int num_blocks = size / BLOCK + 1;
    float_to_bit_gpu_kernel<<<num_blocks, BLOCK, 0, get_cuda_stream()>>>(src, dst, size);
}

// --------------------------------


__device__ __host__ static inline void set_bit(unsigned char *const dst, size_t index) {
    size_t dst_i = index / 8;
    int dst_shift = index % 8;
    dst[dst_i] |= 1 << dst_shift;
    //dst[dst_i] |= 1 << (8 - dst_shift);
}

__device__ __host__ static inline unsigned char get_bit(unsigned char const*const src, size_t index) {
    size_t src_i = index / 8;
    int src_shift = index % 8;
    unsigned char val = (src[src_i] & (1 << src_shift)) > 0;
    //unsigned char val = (src[src_i] & (1 << (8 - src_shift))) > 0;
    return val;
}

// Intel CPUs and nVidia CUDA GPU are little endian
__device__ __host__ unsigned char reverse_byte(unsigned char a)
{
    return ((a & 0x1) << 7) | ((a & 0x2) << 5) |
        ((a & 0x4) << 3) | ((a & 0x8) << 1) |
        ((a & 0x10) >> 1) | ((a & 0x20) >> 3) |
        ((a & 0x40) >> 5) | ((a & 0x80) >> 7);
}

__device__ __host__ unsigned char reverse_byte_2(unsigned char a)
{
    return ((a * 0x0802LU & 0x22110LU) | (a * 0x8020LU & 0x88440LU)) * 0x10101LU >> 16;
}



__device__ __host__ void transpose8rS32_reversed_diagonale(unsigned char* A, int m, int n, unsigned char* B)
{
    unsigned x, y, t;

    // Load the array and pack it into x and y.
    x = (A[0] << 24) | (A[m] << 16) | (A[2 * m] << 8) | A[3 * m];
    y = (A[4 * m] << 24) | (A[5 * m] << 16) | (A[6 * m] << 8) | A[7 * m];

    t = (x ^ (x >> 7)) & 0x00AA00AA;  x = x ^ t ^ (t << 7);
    t = (y ^ (y >> 7)) & 0x00AA00AA;  y = y ^ t ^ (t << 7);

    t = (x ^ (x >> 14)) & 0x0000CCCC;  x = x ^ t ^ (t << 14);
    t = (y ^ (y >> 14)) & 0x0000CCCC;  y = y ^ t ^ (t << 14);

    t = (x & 0xF0F0F0F0) | ((y >> 4) & 0x0F0F0F0F);
    y = ((x << 4) & 0xF0F0F0F0) | (y & 0x0F0F0F0F);
    x = t;

    B[7 * n] = reverse_byte(x >> 24);  B[6 * n] = reverse_byte(x >> 16);  B[5 * n] = reverse_byte(x >> 8);  B[4 * n] = reverse_byte(x);
    B[3 * n] = reverse_byte(y >> 24);  B[2 * n] = reverse_byte(y >> 16);  B[1 * n] = reverse_byte(y >> 8);  B[0 * n] = reverse_byte(y);
}


__global__ void transpose_bin_gpu_kernel(unsigned char *A, unsigned char *B, const int n, const int m,
    const int lda, const int ldb, const int block_size)
{
    int i;
    int index = blockIdx.x*blockDim.x + threadIdx.x;

    //for (i = 0; i < n; i += 8)
    {
        i = (index*8) % n;
        int j;
        //for (j = 0; j < m - 8; j += 8)
        {
            j = ((index * 8) / n) * 8;
            if (j < m - 8) {
                int a_index = i*lda + j;
                int b_index = j*ldb + i;
                //transpose_8x8_bits_my(&A[a_index/8], &B[b_index/8], lda/8, ldb/8);
                transpose8rS32_reversed_diagonale(&A[a_index / 8], lda / 8, ldb / 8, &B[b_index / 8]);
            }
            else if (j < m) {
                for (; j < m; ++j) {
                    if (get_bit(A, i*lda + j)) set_bit(B, j*ldb + i);
                }
            }
        }
    }
}


void transpose_bin_gpu(unsigned char *A, unsigned char *B, const int n, const int m,
    const int lda, const int ldb, const int block_size)
{
    size_t size = n*m/64 + 1;
    const int num_blocks = size / BLOCK + 1;
    transpose_bin_gpu_kernel << <num_blocks, BLOCK, 0, get_cuda_stream() >> >(A, B, n, m, lda, ldb, block_size);
}


// --------------------------------


__global__ void fill_int8_gpu_kernel(unsigned char *src, unsigned char val, size_t size) {
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    if(index < size) src[index] = 0;
}

void fill_int8_gpu(unsigned char *src, unsigned char val, size_t size) {
    const int num_blocks = size / BLOCK + 1;
    fill_int8_gpu_kernel<<<num_blocks, BLOCK, 0, get_cuda_stream() >>>(src, val, size);
}